#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>
#include <exception>
#include <cmath>
#include <iostream>
#include "Matrix.cuh"

bool isCorrectAnswer(const Matrix& m, float val, float eps = 1e-6)
{
    for (size_t i = 0; i < m.hight(); i++) {
        for (size_t j = 0; j < m.width(); j++) {
            if (std::abs(m.at(i, j) - val) > eps)
                return false;
        }
    }
    return true;
}

void printMatrix(const Matrix& m) 
{
    for (size_t i = 0; i < m.hight(); ++i)
    {
        for (size_t j = 0; j < m.width(); ++j)
        {
            std::cout << m.at(i, j) << '\t';
        }
        std::cout << std::endl;
    }
}

int main()
{
    try 
    {
        size_t s = 1 << 10;
        Matrix m1 = Matrix::full(1.f, s*2, s);
        Matrix m2 = Matrix::full(1.f, s, s);
        Matrix m3 = m1.mul(m2, Matrix::MulMode::SHARED);

        if (isCorrectAnswer(m3, static_cast<float>(s)))
            std::cout << "CORRECT";
        else
            std::cout << "WRONG";
        std::cout << " ANSWER" << std::endl;

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            std::cerr << "hipDeviceReset failed!" << std::endl;
            return 1;
        }
    }
    catch (const std::exception& e) 
    {
        std::cerr << e.what() << std::endl;
        return -1;
    }

    return 0;
}
